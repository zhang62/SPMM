#include "hip/hip_runtime.h"
#include "spmm.h"
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
//#include <thrust/execution_policy.h>

/*-----------------------------------------------*/
void cuda_init(int argc, char **argv) {
  int deviceCount, dev;
  hipGetDeviceCount(&deviceCount);
  printf("=========================================\n");
  if (deviceCount == 0)
    printf("There is no device supporting CUDA\n");

  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        printf("There is no device supporting CUDA.\n");
      else if (deviceCount == 1)
        printf("There is 1 device supporting CUDA\n");
      else
        printf("There are %d devices supporting CUDA\n", deviceCount);
    }
    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    printf("  Major revision number:          %d\n",
           deviceProp.major);
    printf("  Minor revision number:          %d\n",
           deviceProp.minor);
    printf("  Total amount of global memory:  %.2f GB\n",
           deviceProp.totalGlobalMem/1e9);
  }
  dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("\nRunning on Device %d: \"%s\"\n", dev, deviceProp.name);
  printf("=========================================\n");
}

/*---------------------------------------------------*/
void cuda_check_err() {
  hipError_t cudaerr = hipGetLastError() ;
  if (cudaerr != hipSuccess)
    printf("error: %s\n",hipGetErrorString(cudaerr));
}

/*
__global__
void csr_spmm_kernel(int m, int k, int n, int *d_ia, int *d_ja, REAL *d_a,
                                          int *d_ib, int *d_jb, REAL *d_b,
                                          int *d_ic, int *d_jc, REAL *d_c)
{
}
*/

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600 && DOUBLEPRECISION == 1
static __inline__ __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

static __device__ __forceinline__ int get_warp_id()
{
    return threadIdx.x >> 5;
}

static __device__ __forceinline__ int get_lane_id()
{
   return threadIdx.x & (WARP-1);
}

static __device__ int HashFunc(int m, int key, int i)
{
   return ((key % m) + i) % m;
}

static __device__ int HashInsert(int   HashSize,      /* capacity of the hash table */
                                 int  *HashKeys,      /* assumed to be initialized as all -1's */
                                 REAL *HashVals,      /* assumed to be initialized as all 0's */
                                 int   key,           /* assumed to be nonnegative */
                                 REAL  val)
{
   int i, ret=-1;
   for (i = 0; i < HashSize; i++)
   {
      /* compute the hash value of key */
      int j = HashFunc(HashSize, key, i);
      /* try to insert key+1 into slot j */
      int old = atomicCAS(HashKeys+j, -1, key);
      if (old == -1 || old == key)
      {
         /* this slot was open or contained 'key', update value */
         atomicAdd(HashVals+j, val);
         ret = j;
         break;
      }
   }
   return ret;
}

__global__
void csr_merge_row(int rowi, int *ia, int *ja, REAL *a,
                             int *ib, int *jb, REAL *b,
                             int *nz, int *ic, REAL *c,
                   int HashSize, int *HashKeys, REAL *HashVals)
{
   int i, j, istart, iend;
   const int NUM_WARPS = BLOCKDIM / WARP;
   const int warp_id = get_warp_id();
   const int lane_id = get_lane_id();

   /*
   int rowA = blockIdx.x * NUM_WARPS + warp_id;
   volatile __shared__ int rownnz[NUM_WARPS];
   rownnz[warp_id] = 0;
   */

   istart = ia[rowi];
   iend = ia[rowi+1];

   for (i = istart + threadIdx.y; i < iend; i += WARP)
   {
      int rowB = ja[i];
      REAL va = a[i];
      for (j = ib[rowB]; j < ib[rowB+1]; j++)
      {
         int pos = HashInsert(HashSize, HashKeys, HashVals, jb[j], va*b[j]);
         assert(pos != -1);
      }
   }
}

void csr_spmm(struct csr_t *A, struct csr_t *B, struct csr_t *C)
{
   int m, k, n, nnzA, nnzB;
   int *d_ia, *d_ja, *d_ib, *d_jb, *d_ic, d_jc;
   REAL *d_a, *d_b, *d_c;
   csr_t C0;

   m = A->nrow;
   k = A->ncol;
   n = B->ncol;
   nnzA = A->nnz;
   nnzB = B->nnz;
   /*---------- Device Memory */
   hipMalloc((void **)&d_ia, (m+1)*sizeof(int));
   hipMalloc((void **)&d_ja,  nnzA*sizeof(int));
   hipMalloc((void **)&d_a,   nnzA*sizeof(REAL));
   hipMalloc((void **)&d_ib, (k+1)*sizeof(int));
   hipMalloc((void **)&d_jb,  nnzB*sizeof(int));
   hipMalloc((void **)&d_b,   nnzB*sizeof(REAL));
   /*---------- Memcpy */
   hipMemcpy(d_ia, A->ia, (m+1)*sizeof(int),  hipMemcpyHostToDevice);
   hipMemcpy(d_ja, A->ja,  nnzA*sizeof(int),  hipMemcpyHostToDevice);
   hipMemcpy(d_a,  A->a,   nnzA*sizeof(REAL), hipMemcpyHostToDevice);
   hipMemcpy(d_ib, B->ia, (k+1)*sizeof(int),  hipMemcpyHostToDevice);
   hipMemcpy(d_jb, B->ja,  nnzB*sizeof(int),  hipMemcpyHostToDevice);
   hipMemcpy(d_b,  B->a,   nnzB*sizeof(REAL), hipMemcpyHostToDevice);

   csr_spmm_cpu(A, B, &C0);
   int rowi = 70;
   int rowi_len = C0.ia[rowi+1] - C0.ia[rowi];

   printf("\n=====\n");
   for (int i = C0.ia[rowi]; i < C0.ia[rowi+1]; i++)
   {
      printf("(%d, %e)\n", C0.ja[i], C0.a[i]);
   }
   printf("\n=====\n");

   int  *d_hash_keys, *h_hash_keys;
   REAL *d_hash_vals, *h_hash_vals;
   hipMalloc((void **)&d_hash_keys, rowi_len*sizeof(int));
   hipMalloc((void **)&d_hash_vals, rowi_len*sizeof(REAL));

   thrust::device_ptr<int> d_hash_keys_ptr(d_hash_keys);
   thrust::fill(d_hash_keys_ptr, d_hash_keys_ptr+rowi_len, -1);
   hipMemset(d_hash_vals, 0, rowi_len*sizeof(REAL));

   dim3 bDim(4,8,1);
   csr_merge_row<<<1,bDim>>>(rowi, d_ia, d_ja, d_a, d_ib, d_jb, d_b, NULL, NULL, NULL,
                             rowi_len, d_hash_keys, d_hash_vals);

   h_hash_keys = (int *)  malloc(rowi_len*sizeof(int));
   h_hash_vals = (REAL *) malloc(rowi_len*sizeof(REAL));
   hipMemcpy(h_hash_keys, d_hash_keys, rowi_len*sizeof(int),  hipMemcpyDeviceToHost);
   hipMemcpy(h_hash_vals, d_hash_vals, rowi_len*sizeof(REAL), hipMemcpyDeviceToHost);
   printf("\n");
   for (int i = 0; i < rowi_len; i++)
   {
      printf("(%d, %e)\n", h_hash_keys[i], h_hash_vals[i]);
   }
   printf("\n");

   hipFree(d_ia);
   hipFree(d_ja);
   hipFree(d_a);
   hipFree(d_ib);
   hipFree(d_jb);
   hipFree(d_b);
   hipFree(d_hash_keys);
   hipFree(d_hash_vals);
   free(h_hash_keys);
   free(h_hash_vals);
}
